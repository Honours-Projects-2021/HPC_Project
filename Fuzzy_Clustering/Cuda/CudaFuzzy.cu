#include "hip/hip_runtime.h"
#include <cmath>
#include <fstream>
#include <iostream>
#include <vector>
#include <stdio.h>
#include<random>
#include <ctime>

#include "./inc/common/book.h"
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include "Data.h"

#define NUMCENTS 3
#define FMEASURE 3
#define EPOCHS  500

using namespace std;

// This function rounds off to 5 decimal places
__device__ double Round(double c){
    return round(c*100000)/100000;
}

void displayCentroids(double* cent, int NumFeatures){
    for(int i = 0; i < NUMCENTS; i++){
        cout << "cluster "<<i+1<<" [ ";
        for(int j = 0; j < NumFeatures; j++){
            if(j != NumFeatures-1)
                printf("%f, ",cent[i*NUMCENTS +j]); 
            else
                printf("%f ]\n",cent[i*NUMCENTS +j]);

        }
    }
}

void displayWeights(double* weights, int n){
    for(int i = 0; i < n; i++){
        cout << "weight for data-point "<<i+1<<" : [ ";
        for(int j = 0; j < NUMCENTS; j++){
            if(j != NUMCENTS-1)
                printf("%f, ",weights[i*NUMCENTS +j]); 
            else
                printf("%f ]\n",weights[i*NUMCENTS +j]);

        }
    }

}


__global__ void computeCentroids(double *data, double *weights, double *centroids, int numCents, int dRows, int dCols , int fMeasure){
   
    int i = threadIdx.x ; // centroid number
    for(int k = i*dCols; k < (i+1)*dCols; k++){
        centroids[k] = 0;
    }

    double denominator = 0;
    
    for(int x = 0; x < dRows; x++){
        double w = pow(weights[x*numCents + i],fMeasure);
        denominator = denominator + w;

        for(int k = i*dCols; k < (i+1)*dCols; k++){
            centroids[k] = centroids[k] + w*data[x*dCols + k];
        }
    }


    for(int k = i*dCols; k < (i+1)*dCols; k++){
        centroids[k] = centroids[k]*(1/denominator);
    }
}


__global__ void computeWeights(double *data, double *weights, double *centroids, int numCents,int dRows, int dCols , int fMeasure){

    int i = threadIdx.x ; // datapoint
    int j = blockIdx.x; // cluster

    double w = 0; // the weight for index [i,j] of the weight matrix
    if(i < dRows)

    // loops for through every centroids
    for(int k = 0; k < numCents; k++){
        double numerator = 0;
        double denominator = 0;

        // calculates the distances between the data points and centroids as described in the formula
        for(int l = 0; l < dCols; l++){
            double dPoint = data[i*dCols+l]; // the data point we are dealing with
            numerator += pow((dPoint - centroids[j*dCols+l]),2);
            denominator += pow((dPoint - centroids[k*dCols+l]),2);
            
        }
        numerator = sqrt(numerator); // top distance
        denominator = sqrt(denominator); // bottom distance
        w += pow((numerator/denominator),(2/(fMeasure-1))); // add to w
    }

    if(i < dRows)
    weights[i*numCents + j] = 1/w; // Put the weight into the weight matrix
}

// sets every value in the array to zero
void initCentroids(double *cents, int numCols){
    for(int i = 0; i < numCols*NUMCENTS; i++ ){
        cents[i] = 0.0;
    }
}

// creates an array equivalent of the given vector
void initWeights(vector<double> flatWeights, double *weights){
    for(int i = 0; i < flatWeights.size(); i++){
        weights[i] = flatWeights.at(i);
    }
}
int main(){

    // Import in the data
    Data d = Data("../Utils/wine-clustering.csv"); // The complete datasets of 13 columns and 178 records
    Data w = Data("../Utils/weights.csv");  // The initialized weight matrix

    // Attributes of the dataset and the weight matrix
    int dataRows = d.getNumRows();
    int dataColumns = d.getNumCols();
    int weightRows = w.getNumRows();
    int weightCols = w.getNumCols();

    // Size variables for hipMalloc funcntions
    int centSize = NUMCENTS*dataColumns*sizeof(double); // Number of centroids
    int dataSize = dataRows*dataColumns*sizeof(double); // Number of data points
    int weightSize = dataRows*NUMCENTS*sizeof(double);  // Size of the weight matrix


    // Flattened dataset and weights to make it easy for us to work with
    vector<double> flatData = d.getFlat(); // flattened data
    vector<double> flatWeights = w.getFlat(); // flattened weights

    // Create centroids arrays and weights arrays
    double *centroids = (double*) malloc(NUMCENTS*dataColumns*sizeof(double)); // flattened centroids
    double *weights = (double*) malloc(dataRows*dataColumns*sizeof(double)); // array equivalent of our vector weights array


    initWeights(flatWeights , weights); // converts vector into a normal array
    initCentroids(centroids , dataColumns); // initialize all array values to 0's

    // data for device declaration
    double *deviceWeights, *deviceData , *deviceCentroids;

    // allocate space in the device
    checkCudaErrors(hipMalloc((void**) &deviceData, dataSize));
    checkCudaErrors(hipMalloc((void**) &deviceWeights, weightSize));
    checkCudaErrors(hipMalloc((void**) &deviceCentroids, centSize));

    // copying memory
    checkCudaErrors(hipMemcpy(deviceData , &flatData[0] , dataSize, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(deviceWeights , weights , weightSize, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(deviceCentroids , centroids, centSize, hipMemcpyHostToDevice));

    // Number of blocks and threads for calcualting the centroids
    dim3 Cblock(1);
    dim3 Cthreads(NUMCENTS);
    
    // Number of blocks and threads for calculating the weights
    dim3 Wblock(NUMCENTS);
    dim3 Wthreads(dataRows);


    StopWatchInterface *se_timer = NULL;
    sdkCreateTimer(&se_timer);
    sdkStartTimer(&se_timer);


    // Run the algorithm for the number of epochs
    for(int i = 0; i < EPOCHS; i++){
        // Compute the new centroids
        computeCentroids<<<Cblock,Cthreads>>>(deviceData, deviceWeights, deviceCentroids, NUMCENTS, dataRows, dataColumns , FMEASURE);
        // Compute the new weights
        computeWeights<<<Wblock,Wthreads>>>(deviceData, deviceWeights, deviceCentroids, weightCols, dataRows, dataColumns , FMEASURE);
    }
    

    sdkStopTimer(&se_timer);
    printf("Processing time for Cuda Parallel: %f (ms)\n", sdkGetTimerValue(&se_timer));
    sdkDeleteTimer(&se_timer);

    // copy the results into the respective arrays
    checkCudaErrors(hipMemcpy(centroids, deviceCentroids ,  centSize, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(weights, deviceWeights ,  weightSize, hipMemcpyDeviceToHost));


    // for(int i = 0; i < 100; i++){
    //     for(int j = 0; j < 3; j++){
    //         printf("%.5f " ,weights[i*NUMCENTS + j]);
    //     }
    //     printf("\n");
    // }
    // printf("\n");
    
    displayCentroids(centroids,dataColumns);
    // free acquried Device memory
    checkCudaErrors(hipFree(deviceCentroids));
    checkCudaErrors(hipFree(deviceWeights));
    checkCudaErrors(hipFree(deviceData));

    // Free acquired Host memory
    free(centroids);
    free(weights);


    return 0;
}




